#include "hip/hip_runtime.h"
#include <stdio.h>
#include <malloc.h>
#include "Turbo.h"
/*library of GPU*/
#include "math_functions.hpp"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include "hip/hip_texture_types.h"

/*pre-set parameters chosed with CQI*/
#define win_N  24						//inner Parallel Sliding Window

#define win_L  100
#define half_win_L 50
#define info_L  2400
#define info_tL 2432
#define block_N 44
#define ab_thread_N 384
#define half_abt_N 192
#define info_thread_N 400
#define info_thread_tN 403
#define cal_time 6
#define bank_size 32

/*
���������
dev_gamma��		gamma��

���������
dev_a��			������Ϣ��
dev_s��			ϵͳ��Ϣ��
dev_p��			У����Ϣ��
*/
__global__ void extMapKernel(float *dev_llr, float *dev_a, float *dev_s, int *dev_inter, int interleave_type)
{
	__shared__ float shr_a[info_L];

	unsigned int a_idx, s_idx, i_idx, i;
	i_idx = threadIdx.x;
	a_idx = blockIdx.x*info_L;
	s_idx = blockIdx.x*info_tL;

	if (interleave_type == 0)
	{
		for (i = 0; i < cal_time; i++)
		{
			shr_a[i_idx] = dev_llr[a_idx + i_idx] - dev_s[s_idx + i_idx] - dev_a[a_idx + i_idx];
			i_idx += info_thread_N;
		}
		__syncthreads();
		for (i = 0; i < cal_time; i++)
		{
			i_idx -= info_thread_N;
			dev_a[a_idx + i_idx] = shr_a[dev_inter[i_idx]];
		}
	}
	else
	{
		for (i = 0; i < cal_time; i++)
		{
			shr_a[dev_inter[i_idx]] = dev_llr[a_idx + i_idx] - dev_s[s_idx + i_idx] - dev_a[a_idx + i_idx];
			i_idx += info_thread_N;
		}
		__syncthreads();
		for (i = 0; i < cal_time; i++)
		{
			i_idx -= info_thread_N;
			dev_a[a_idx + i_idx] = shr_a[i_idx];
		}
	}
}
__global__ void abMapKernel(float *dev_llr, float *dev_gamma, float *dev_ab, float *last_alfa, float *last_beta, int *dev_para, int iteration) //last iteration value
{
	__shared__ float shr_8illr[half_abt_N << 3];
	__shared__ float shr_8jllr[half_abt_N << 3];
	float plus, minus, gamma, ab;

	unsigned int half_idx = threadIdx.x%half_abt_N;
	unsigned int win_n = half_idx >> 3;
	unsigned int state_n = half_idx & 7;
	unsigned int i;
	int add_loc, sub_loc;
	float *g_ptr, *ab_ptr;

	if (threadIdx.x < half_abt_N)
	{
		/*get parameter from mem*/
		ab_ptr = dev_ab + ((blockIdx.x*info_L << 3) + half_idx);
		g_ptr = dev_gamma + (((blockIdx.x << 1) + dev_para[state_n])*info_tL + win_n*win_L);
		add_loc = dev_para[16 + state_n];
		sub_loc = dev_para[32 + state_n];
		/*get last alfa*/
		if (iteration == 0)
		{
			ab = state_n == 0 ? 0.0f : -10000.0f;

			if (win_n > 0)
			{
				for (i = 3; i >0; i--)
				{
					gamma = *(g_ptr - i);
					plus = __shfl(ab, add_loc, 8) + gamma;
					minus = __shfl(ab, sub_loc, 8) - gamma;
					ab = logf(expf(plus) + expf(minus));
				}
			}
		}
		else
		{
			ab = last_alfa[blockIdx.x*half_abt_N + half_idx];
		}

		/*calculate half alfa*/
		for (i = 0; i < half_win_L; i++)
		{
			*ab_ptr = ab;
			gamma = *g_ptr;
			plus = __shfl(ab, add_loc, 8) + gamma;
			minus = __shfl(ab, sub_loc, 8) - gamma;
			ab = logf(expf(plus) + expf(minus));
			ab_ptr += half_abt_N;
			g_ptr++;
		}
	}
	else
	{
		/*get parameter from mem*/
		ab_ptr = dev_ab + ((blockIdx.x*info_L << 3) + (win_L - 1)*half_abt_N + half_idx);
		g_ptr = dev_gamma + (((blockIdx.x << 1) + dev_para[8 + state_n])*info_tL + win_n*win_L + win_L - 1);
		add_loc = dev_para[24 + state_n];
		sub_loc = dev_para[40 + state_n];

		/*get last beta*/
		if (iteration == 0)
		{
			int v_L = win_n == win_N - 1 ? 3 : 20;
			ab = state_n == 0 ? 0.0f : -10000.0f;
			for (i = v_L; i >0; i--)
			{
				gamma = *(g_ptr + i);
				plus = __shfl(ab, add_loc, 8) + gamma;
				minus = __shfl(ab, sub_loc, 8) - gamma;
				ab = logf(expf(plus) + expf(minus));
			}
			if (win_n == win_N - 1)
			{
				last_beta[blockIdx.x*half_abt_N + half_idx] = ab;
			}
		}
		else
		{
			ab = last_beta[blockIdx.x*half_abt_N + half_idx];
		}

		/*calculate half beta*/
		for (i = 0; i < half_win_L; i++)
		{
			*ab_ptr = ab;
			gamma = *g_ptr;
			plus = __shfl(ab, add_loc, 8) + gamma;
			minus = __shfl(ab, sub_loc, 8) - gamma;
			ab = logf(expf(plus) + expf(minus));

			ab_ptr -= half_abt_N;
			g_ptr--;
		}
	}
	__syncthreads();
	/*calculate rest alfa\beta and prepare for llr*/

	if (threadIdx.x < half_abt_N)
	{
		float minus_tmp, plus_tmp;
		float *a_ptr = dev_llr + (blockIdx.x*info_L + win_n* win_L + half_win_L + state_n);
		float *shr_ptr = shr_8illr + ((win_n << 3) + state_n*half_abt_N);
		unsigned int tmp_loc = state_n & 3;

		for (int count = 0; i < win_L; i++, count++)
		{
			gamma = *g_ptr;
			plus = __shfl(ab, add_loc, 8) + gamma;
			minus = __shfl(ab, sub_loc, 8) - gamma;
			ab = logf(expf(plus) + expf(minus));

			minus = *ab_ptr + minus;
			plus = *ab_ptr + plus;

			minus_tmp = __shfl_down(minus, 4, 8);
			plus_tmp = __shfl_up(plus, 4, 8);
			if (state_n <4)
			{
				shr_8illr[count*half_abt_N + half_idx] = logf(expf(minus_tmp) + expf(minus));
			}
			else
			{
				shr_8illr[count*half_abt_N + half_idx] = logf(expf(plus_tmp) + expf(plus));
			}

			if (count == 7 || i == win_L - 1 && state_n<(half_win_L & 7))
			{
				count = -1;

				*a_ptr = logf(expf(*(shr_ptr + 4 + tmp_loc)) + expf(*(shr_ptr + 4 + (tmp_loc + 1 & 3))) + expf(*(shr_ptr + 4 + (tmp_loc + 2 & 3))) + expf(*(shr_ptr + 4 + (tmp_loc + 3 & 3)))) - logf(expf(*(shr_ptr + tmp_loc)) + expf(*(shr_ptr + (tmp_loc + 1 & 3))) + expf(*(shr_ptr + (tmp_loc + 2 & 3))) + expf(*(shr_ptr + (tmp_loc + 3 & 3))));
				a_ptr += 8;
			}
			ab_ptr += half_abt_N;
			g_ptr++;
		}
		if (win_n < win_N - 1)
		{
			last_alfa[blockIdx.x*half_abt_N + half_idx + 8] = ab;
		}
	}
	else
	{
		float minus_tmp, plus_tmp;
		float *a_ptr = dev_llr + (blockIdx.x*info_L + win_n* win_L + half_win_L - 1 - state_n);
		float *shr_ptr = shr_8jllr + ((win_n << 3) + state_n*half_abt_N);
		unsigned int tmp_loc = state_n & 3;

		for (int count = 0; i < win_L; i++, count++)
		{
			gamma = *g_ptr;
			plus = __shfl(ab, add_loc, 8) + gamma;
			minus = __shfl(ab, sub_loc, 8) - gamma;
			ab = logf(expf(plus) + expf(minus));

			minus = *ab_ptr + minus;
			plus = *ab_ptr + plus;

			minus_tmp = __shfl_down(minus, 4, 8);
			plus_tmp = __shfl_up(plus, 4, 8);
			if (state_n <4)
			{
				shr_8jllr[count*half_abt_N + half_idx] = logf(expf(minus_tmp) + expf(minus));
			}
			else
			{
				shr_8jllr[count*half_abt_N + half_idx] = logf(expf(plus_tmp) + expf(plus));
			}

			if (count == 7 || i == win_L - 1 && state_n<(half_win_L & 7))
			{
				count = -1;
				*a_ptr = logf(expf(*(shr_ptr + 4 + tmp_loc)) + expf(*(shr_ptr + 4 + (tmp_loc + 1 & 3))) + expf(*(shr_ptr + 4 + (tmp_loc + 2 & 3))) + expf(*(shr_ptr + 4 + (tmp_loc + 3 & 3)))) - logf(expf(*(shr_ptr + tmp_loc)) + expf(*(shr_ptr + (tmp_loc + 1 & 3))) + expf(*(shr_ptr + (tmp_loc + 2 & 3))) + expf(*(shr_ptr + (tmp_loc + 3 & 3))));
				a_ptr -= 8;
			}
			ab_ptr -= half_abt_N;
			g_ptr--;
		}
		if (win_n > 0)
		{
			last_beta[blockIdx.x*half_abt_N + half_idx - 8] = ab;
		}
	}
}

__global__ void gammaKernel(float *dev_gamma, float *dev_s, float *dev_p, float *dev_a)
{
	float gamma, p;
	unsigned g_idx, a_idx, s_idx, i;

	if (threadIdx.x < info_thread_N)
	{
		s_idx = blockIdx.x*info_tL + threadIdx.x;
		g_idx = (blockIdx.x*info_tL << 1) + threadIdx.x;
		a_idx = blockIdx.x*info_L + threadIdx.x;

		if (dev_a != NULL)
		{
			for (i = 0; i < cal_time; i++)
			{
				p = dev_p[s_idx];
				gamma = (dev_s[s_idx] - p + dev_a[a_idx])*0.5f;
				dev_gamma[g_idx] = gamma;
				dev_gamma[g_idx + info_tL] = gamma + p;
				s_idx += info_thread_N;
				a_idx += info_thread_N;
				g_idx += info_thread_N;
			}
		}
		else
		{
			for (i = 0; i < cal_time; i++)
			{
				p = dev_p[s_idx];
				gamma = (dev_s[s_idx] - p)*0.5f;
				dev_gamma[g_idx] = gamma;
				dev_gamma[g_idx + info_tL] = gamma + p;
				s_idx += info_thread_N;
				g_idx += info_thread_N;
			}
		}
	}
	else
	{
		s_idx = blockIdx.x*info_tL + info_L + threadIdx.x - info_thread_N;
		g_idx = (blockIdx.x*info_tL<<1) + info_L + threadIdx.x - info_thread_N;

		p = dev_p[s_idx];
		gamma = (dev_s[s_idx] - p)*0.5f;
		dev_gamma[g_idx] = gamma;
		dev_gamma[g_idx + info_tL] = gamma + p;
	}
}

__global__ void extKernel(float *dev_llr, float *dev_a, float *dev_s, int *dev_inter, int interleave_type)
{
	__shared__ float shr_a[info_L];

	unsigned int a_idx, s_idx, i_idx;
	i_idx = threadIdx.x;
	a_idx = blockIdx.x*info_L;
	s_idx = blockIdx.x*info_tL;

	if (interleave_type == 0)
	{
		for (i_idx = threadIdx.x; i_idx < info_L; i_idx += info_thread_N)
		{
			shr_a[i_idx] = 0.7f*(dev_llr[a_idx + i_idx] - dev_s[s_idx + i_idx] - dev_a[a_idx + i_idx]);
			
		}
		__syncthreads();
		for (i_idx = threadIdx.x; i_idx < info_L; i_idx += info_thread_N)
		{			
			dev_a[a_idx + i_idx] = shr_a[dev_inter[i_idx]];
		}
	}
	else
	{
		for (i_idx = threadIdx.x; i_idx < info_L; i_idx += info_thread_N)
		{
			shr_a[dev_inter[i_idx]] = 0.7f*(dev_llr[a_idx + i_idx] - dev_s[s_idx + i_idx] - dev_a[a_idx + i_idx]);
		}
		__syncthreads();
		for (i_idx = threadIdx.x; i_idx < info_L; i_idx += info_thread_N)
		{
			dev_a[a_idx + i_idx] = shr_a[i_idx];
		}
	}
}

__global__ void ab2Kernel(float *dev_llr, float *dev_gamma, float *dev_ab, float *last_alfa, float *last_beta, int *dev_para, int iteration) //last iteration value
{
	__shared__ float shr_8illr[half_abt_N << 3];
	__shared__ float shr_8jllr[half_abt_N << 3];

	float plus, minus, gamma, ab;

	unsigned int half_idx = threadIdx.x%half_abt_N;
	unsigned int win_n = half_idx >> 3;
	unsigned int state_n = half_idx & 7;
	unsigned int i;
	int add_loc, sub_loc;
	float *g_ptr, *ab_ptr;

	//parameter set and pretrain
	if (threadIdx.x < half_abt_N)
	{
		/*get parameter from mem*/
		ab_ptr = dev_ab + ((blockIdx.x*info_L << 3) + half_idx);
		g_ptr = dev_gamma + (((blockIdx.x << 1) + dev_para[state_n])*info_tL + win_n*win_L);
		add_loc = dev_para[16 + state_n];
		sub_loc = dev_para[32 + state_n];
		/*get last alfa*/
		if (iteration == 0)
		{
			ab = state_n == 0 ? 0.0f : -10000.0f;

			if (win_n > 0)
			{
				for (i = 20; i >0; i--)
				{
					gamma = *(g_ptr - i);
					plus = __shfl(ab, add_loc, 8) + gamma;
					minus = __shfl(ab, sub_loc, 8) - gamma;
					ab = fmaxf(plus, minus);
				}
			}
		}
		else
		{
			ab = last_alfa[(blockIdx.x*half_abt_N) + half_idx];
		}	
	}
	else
	{
		/*get parameter from mem*/
		ab_ptr = dev_ab + ((blockIdx.x*info_L << 3) + (win_L - 1)*half_abt_N + half_idx);
		g_ptr = dev_gamma + (((blockIdx.x << 1) + dev_para[8 + state_n])*info_tL + win_n*win_L + win_L - 1);
		add_loc = dev_para[24 + state_n];
		sub_loc = dev_para[40 + state_n];

		/*get last beta*/
		if (iteration == 0)
		{
			int v_L = win_n == win_N - 1 ? 3 : 20;
			ab = state_n == 0 ? 0.0f : -10000.0f;
			for (i = v_L; i >0; i--)
			{
				gamma = *(g_ptr + i);
				plus = __shfl(ab, add_loc, 8) + gamma;
				minus = __shfl(ab, sub_loc, 8) - gamma;
				ab = fmaxf(plus, minus);
			}
			if (win_n == win_N - 1)
			{
				last_beta[(blockIdx.x*half_abt_N) + half_idx] = ab;
			}
		}
		else
		{
			ab = last_beta[(blockIdx.x*half_abt_N) + half_idx];
		}		
	}

	//calculate half alfa beta
	if (threadIdx.x < half_abt_N)
	{
		/*calculate half alfa*/
		for (i = 0; i < half_win_L; i++)
		{
			*ab_ptr = ab;
			gamma = *g_ptr;
			plus = __shfl(ab, add_loc, 8) + gamma;
			minus = __shfl(ab, sub_loc, 8) - gamma;
			ab = fmaxf(plus, minus);
			ab_ptr += half_abt_N;
			g_ptr++;
			
		}
	}
	else
	{
		/*calculate half beta*/
		for (i = 0; i < half_win_L; i++)
		{
			*ab_ptr = ab;
			gamma = *g_ptr;
			plus = __shfl(ab, add_loc, 8) + gamma;
			minus = __shfl(ab, sub_loc, 8) - gamma;
			ab = fmaxf(plus, minus);
			ab_ptr -= half_abt_N;
			g_ptr--;
		}
	}
	__syncthreads();
	/*calculate rest alfa\beta and prepare for llr*/

	if (threadIdx.x < half_abt_N)
	{
		float minus_tmp, plus_tmp;
		float *a_ptr = dev_llr + (blockIdx.x*info_L + win_n* win_L + half_win_L + state_n);
		float *shr_ptr = shr_8illr + ((win_n << 3) + state_n*half_abt_N);
		unsigned int tmp_loc = state_n & 3;

		for (int count = 0; i < win_L; i++, count++)
		{
			gamma = *g_ptr;
			plus = __shfl(ab, add_loc, 8) + gamma;
			minus = __shfl(ab, sub_loc, 8) - gamma;
			ab = fmaxf(plus, minus);

			minus = *ab_ptr + minus;
			plus = *ab_ptr + plus;

			minus_tmp = __shfl_down(minus, 4, 8);
			plus_tmp = __shfl_up(plus, 4, 8);
			if (state_n <4)
			{
				shr_8illr[count*half_abt_N + half_idx] = fmaxf(minus_tmp, minus);
			}
			else
			{
				shr_8illr[count*half_abt_N + half_idx] = fmaxf(plus_tmp, plus);
			}

			if (count == 7 || i == win_L - 1 && state_n<(half_win_L & 7))
			{
				count = -1;

				*a_ptr = fmaxf(fmaxf(*(shr_ptr + 4 + tmp_loc), *(shr_ptr + 4 + (tmp_loc + 1 & 3))), fmaxf(*(shr_ptr + 4 + (tmp_loc + 2 & 3)), *(shr_ptr + 4 + (tmp_loc + 3 & 3)))) - fmaxf(fmaxf(*(shr_ptr + tmp_loc), *(shr_ptr + (tmp_loc + 1 & 3))), fmaxf(*(shr_ptr + (tmp_loc + 2 & 3)), *(shr_ptr + (tmp_loc + 3 & 3))));
				a_ptr += 8;
			}
			ab_ptr += half_abt_N;
			g_ptr++;
		}
		if (win_n < win_N - 1)
		{
			last_alfa[blockIdx.x*half_abt_N + half_idx + 8] = ab;
		}
	}
	else
	{
		float minus_tmp, plus_tmp;
		float *a_ptr = dev_llr + (blockIdx.x*info_L + win_n* win_L + half_win_L - 1 - state_n);
		float *shr_ptr = shr_8jllr + ((win_n << 3) + state_n*half_abt_N);
		unsigned int tmp_loc = state_n & 3;

		for (int count = 0; i < win_L; i++, count++)
		{
			gamma = *g_ptr;
			plus = __shfl(ab, add_loc, 8) + gamma;
			minus = __shfl(ab, sub_loc, 8) - gamma;
			ab = fmaxf(plus, minus);

			minus = *ab_ptr + minus;
			plus = *ab_ptr + plus;

			minus_tmp = __shfl_down(minus, 4, 8);
			plus_tmp = __shfl_up(plus, 4, 8);
			if (state_n <4)
			{
				shr_8jllr[count*half_abt_N + half_idx] = fmaxf(minus_tmp, minus);
			}
			else
			{
				shr_8jllr[count*half_abt_N + half_idx] = fmaxf(plus_tmp, plus);
			}

			if (count == 7 || i == win_L - 1 && state_n<(half_win_L & 7))
			{
				count = -1;

				*a_ptr = fmaxf(fmaxf(*(shr_ptr + 4 + tmp_loc), *(shr_ptr + 4 + (tmp_loc + 1 & 3))), fmaxf(*(shr_ptr + 4 + (tmp_loc + 2 & 3)), *(shr_ptr + 4 + (tmp_loc + 3 & 3)))) - fmaxf(fmaxf(*(shr_ptr + tmp_loc), *(shr_ptr + (tmp_loc + 1 & 3))), fmaxf(*(shr_ptr + (tmp_loc + 2 & 3)), *(shr_ptr + (tmp_loc + 3 & 3))));
				a_ptr -= 8;
			}
			ab_ptr -= half_abt_N;
			g_ptr--;
		}
		if (win_n > 0)
		{
			last_beta[blockIdx.x*half_abt_N + half_idx - 8] = ab;
		}
	}
}
__global__ void abKernel(float *dev_llr, float *dev_gamma, float *dev_ab, float *last_alfa, float *last_beta, int *dev_para, int iteration) //last iteration value
{
	__shared__ float shr_8illr[33 * 6 << 3];
	__shared__ float shr_8jllr[33 * 6 << 3];

	float plus, minus, gamma, ab;

	unsigned int half_idx = threadIdx.x%half_abt_N;
	unsigned int win_n = half_idx >> 3;
	unsigned int state_n = half_idx & 7;
	unsigned int i;
	int add_loc, sub_loc;
	float *g_ptr, *ab_ptr;

	//parameter set and pretrain
	if (threadIdx.x < half_abt_N)
	{
		/*get parameter from mem*/
		ab_ptr = dev_ab + ((blockIdx.x*info_L << 3) + half_idx);
		g_ptr = dev_gamma + (((blockIdx.x << 1) + dev_para[state_n])*info_tL + win_n*win_L);
		add_loc = dev_para[16 + state_n];
		sub_loc = dev_para[32 + state_n];
		/*get last alfa*/
		if (iteration == 0)
		{
			ab = state_n == 0 ? 0.0f : -10000.0f;

			if (win_n > 0)
			{
				for (i = 20; i >0; i--)
				{
					gamma = *(g_ptr - i);
					plus = __shfl(ab, add_loc, 8) + gamma;
					minus = __shfl(ab, sub_loc, 8) - gamma;
					ab = fmaxf(plus, minus);
				}
			}
		}
		else
		{
			ab = last_alfa[(blockIdx.x*half_abt_N) + half_idx];
		}
	}
	else
	{
		/*get parameter from mem*/
		ab_ptr = dev_ab + ((blockIdx.x*info_L << 3) + (win_L - 1)*half_abt_N + half_idx);
		g_ptr = dev_gamma + (((blockIdx.x << 1) + dev_para[8 + state_n])*info_tL + win_n*win_L + win_L - 1);
		add_loc = dev_para[24 + state_n];
		sub_loc = dev_para[40 + state_n];

		/*get last beta*/
		if (iteration == 0)
		{
			int v_L = win_n == win_N - 1 ? 3 : 20;
			ab = state_n == 0 ? 0.0f : -10000.0f;
			for (i = v_L; i >0; i--)
			{
				gamma = *(g_ptr + i);
				plus = __shfl(ab, add_loc, 8) + gamma;
				minus = __shfl(ab, sub_loc, 8) - gamma;
				ab = fmaxf(plus, minus);
			}
			if (win_n == win_N - 1)
			{
				last_beta[(blockIdx.x*half_abt_N) + half_idx] = ab;
			}
		}
		else
		{
			ab = last_beta[(blockIdx.x*half_abt_N) + half_idx];
		}
	}

	//calculate half alfa beta
	if (threadIdx.x < half_abt_N)
	{
		/*calculate half alfa*/
		for (i = 0; i < half_win_L; i++)
		{
			*ab_ptr = ab;
			gamma = *g_ptr;
			plus = __shfl(ab, add_loc, 8) + gamma;
			minus = __shfl(ab, sub_loc, 8) - gamma;
			ab = fmaxf(plus, minus);
			ab_ptr += half_abt_N;
			g_ptr++;

		}
	}
	else
	{
		/*calculate half beta*/
		for (i = 0; i < half_win_L; i++)
		{
			*ab_ptr = ab;
			gamma = *g_ptr;
			plus = __shfl(ab, add_loc, 8) + gamma;
			minus = __shfl(ab, sub_loc, 8) - gamma;
			ab = fmaxf(plus, minus);
			ab_ptr -= half_abt_N;
			g_ptr--;
		}
	}
	__syncthreads();
	/*calculate rest alfa\beta and prepare for llr*/

	if (threadIdx.x < half_abt_N)
	{
		float minus_tmp, plus_tmp;
		float *a_ptr = dev_llr + (blockIdx.x*info_L + win_n* win_L + half_win_L + state_n);
		float *shr_ptr = shr_8illr + ((win_n << 3) + (win_n >> 2) + state_n*33 * 6);
		//unsigned int tmp_loc = state_n & 3;

		for (int count = 0; i < win_L; i++, count++)
		{
			gamma = *g_ptr;
			plus = __shfl(ab, add_loc, 8) + gamma;
			minus = __shfl(ab, sub_loc, 8) - gamma;
			ab = fmaxf(plus, minus);

			minus = *ab_ptr + minus;
			plus = *ab_ptr + plus;

			minus_tmp = __shfl_down(minus, 4, 8);
			plus_tmp = __shfl_up(plus, 4, 8);
			if (state_n <4)
			{
				shr_8illr[count*33 * 6 + half_idx + (win_n >> 2)] = fmaxf(minus_tmp, minus);
			}
			else
			{
				shr_8illr[count*33 * 6 + half_idx + (win_n >> 2)] = fmaxf(plus_tmp, plus);
			}

			if (count == 7 || i == win_L - 1 && state_n<(half_win_L & 7))
			{
				count = -1;

				*a_ptr = fmaxf(fmaxf(*(shr_ptr + 7), *(shr_ptr + 6)), fmaxf(*(shr_ptr + 5), *(shr_ptr + 4 ))) - fmaxf(fmaxf(*(shr_ptr + 3), *(shr_ptr + 2)), fmaxf(*(shr_ptr + 1), *shr_ptr));
				a_ptr += 8;
			}
			ab_ptr += half_abt_N;
			g_ptr++;
		}
		if (win_n < win_N - 1)
		{
			last_alfa[blockIdx.x*half_abt_N + half_idx + 8] = ab;
		}
	}
	else
	{
		float minus_tmp, plus_tmp;
		float *a_ptr = dev_llr + (blockIdx.x*info_L + win_n* win_L + half_win_L - 1 - state_n);
		float *shr_ptr = shr_8jllr + ((win_n << 3) + (win_n >> 2) + state_n * 33 * 6);
		//unsigned int tmp_loc = state_n & 3;

		for (int count = 0; i < win_L; i++, count++)
		{
			gamma = *g_ptr;
			plus = __shfl(ab, add_loc, 8) + gamma;
			minus = __shfl(ab, sub_loc, 8) - gamma;
			ab = fmaxf(plus, minus);

			minus = *ab_ptr + minus;
			plus = *ab_ptr + plus;

			minus_tmp = __shfl_down(minus, 4, 8);
			plus_tmp = __shfl_up(plus, 4, 8);
			if (state_n <4)
			{
				shr_8jllr[count * 33 * 6 + half_idx + (win_n >> 2)] = fmaxf(minus_tmp, minus);
			}
			else
			{
				shr_8jllr[count * 33 * 6 + half_idx + (win_n >> 2)] = fmaxf(plus_tmp, plus);
			}

			if (count == 7 || i == win_L - 1 && state_n<(half_win_L & 7))
			{
				count = -1;

				*a_ptr = fmaxf(fmaxf(*(shr_ptr + 7), *(shr_ptr + 6)), fmaxf(*(shr_ptr + 5), *(shr_ptr + 4))) - fmaxf(fmaxf(*(shr_ptr + 3), *(shr_ptr + 2)), fmaxf(*(shr_ptr + 1), *shr_ptr));
				a_ptr -= 8;
			}
			ab_ptr -= half_abt_N;
			g_ptr--;
		}
		if (win_n > 0)
		{
			last_beta[blockIdx.x*half_abt_N + half_idx - 8] = ab;
		}
	}
}

__global__ void dataKernel(unsigned char *dev_data, float *dev_llr, int *dev_inter)
{
	__shared__ float shr_data[info_L];
	unsigned int i_idx, a_base, a_idx;
	
	a_idx = blockIdx.x*info_L;
	for (i_idx = threadIdx.x; i_idx < info_L; i_idx += info_thread_N)
	{
		shr_data[dev_inter[i_idx]] = dev_llr[a_idx + i_idx];	
	}
	__syncthreads();
	
	for (i_idx = threadIdx.x; i_idx < info_L; i_idx += info_thread_N)
	{
		dev_data[a_idx + i_idx] = shr_data[i_idx] > 0 ? 1 : 0;
		
	}
}
__global__ void data2Kernel(unsigned char *dev_data, float *dev_llr, int *dev_inter)
{
	__shared__ float shr_data[info_L];
	unsigned int i_idx, a_base, a_idx, i;
	i_idx = threadIdx.x;
	a_idx = blockIdx.x*info_L;
	for (i = 0; i < cal_time; i++)
	{
		shr_data[dev_inter[i_idx]] = dev_llr[a_idx + i_idx];
		i_idx += info_thread_N;
	}
	__syncthreads();
	for (i = 0; i < cal_time; i++)
	{
		i_idx -= info_thread_N;
		dev_data[a_idx + i_idx] = shr_data[i_idx] > 0 ? 1 : 0;

	}
}
/*
���������
detected_data��		���������Ϣ���أ�

���������
info_L��			��Ϣ���س��ȣ�
input_c_fix��		����Turbo�룻
turbo_code_L��	�����Turbo�볤�ȣ�
interleaver_table��	��֯��
CQI					�ŵ�����ָʾ
*/
float cuTurboDecode(
	unsigned char  *detected_data,
	const float *input_c,
	const int *interleaver_table,
	int info_len,
	int turbo_code_L,
	int CQI)
{
	hipEvent_t     start, stop;
	float duration;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	/*check device*/
	hipDeviceProp_t  prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);
	if (!prop.deviceOverlap)
	{
		printf("Device will not handle overlaps, so no speed up from streams\n");
	}

	/*build mem for detected data in GPU*/
	unsigned char *dev_data;
	hipMalloc((void**)&dev_data, block_N * info_len*sizeof(unsigned char));
	/*build mem for alfa and beta*/
	float *dev_ab;
	hipMalloc((void**)&dev_ab, block_N * info_len * 8 * sizeof(float));//aaaa...bbb...aaa...bbb...
	/*gamma in device*/
	float *dev_gamma;
	hipMalloc((void**)&dev_gamma, block_N * info_tL * 2 * sizeof(float));
	float *dev_a;
	hipMalloc((void**)&dev_a, block_N * info_len * sizeof(float));

	float *dev_llr;
	hipMalloc((void**)&dev_llr, block_N * info_len  * sizeof(float));

	float *last_alfa0, *last_beta0, *last_alfa1, *last_beta1;
	hipMalloc((void**)&last_alfa0, block_N * win_N * 8 * sizeof(float));
	hipMalloc((void**)&last_beta0, block_N * win_N * 8 * sizeof(float));
	hipMalloc((void**)&last_alfa1, block_N * win_N * 8 * sizeof(float));
	hipMalloc((void**)&last_beta1, block_N * win_N * 8 * sizeof(float));
	float last_init[8] = { 0, -100, -100, -100, -100, -100, -100, -100 };
	for (int i = 0; i < block_N; i++)
	{
		hipMemcpy(last_alfa0 + i * win_N * 8, last_init, 8 * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(last_alfa1 + i * win_N * 8, last_init, 8 * sizeof(float), hipMemcpyHostToDevice);
	}

	/*copy interleave table from host to device*/
	int *dev_inter;
	hipMalloc((void**)&dev_inter, info_len*sizeof(int));
	hipMemcpy(dev_inter, interleaver_table, info_len*sizeof(int), hipMemcpyHostToDevice);

	int *dev_para;
	hipMalloc((void**)&dev_para, 48 * sizeof(int));
	int tmp[48] = { 1, 0, 0, 1, 1, 0, 0, 1, 1, 1, 0, 0, 0, 0, 1, 1,
		1, 2, 5, 6, 0, 3, 4, 7, 4, 0, 1, 5, 6, 2, 3, 7,
		0, 3, 4, 7, 1, 2, 5, 6, 0, 4, 5, 1, 2, 6, 7, 3 };
	hipMemcpy(dev_para, tmp, 48 * sizeof(int), hipMemcpyHostToDevice);

	float *dev_s0;
	float *dev_p0;
	float *dev_s1;
	float *dev_p1;

	hipMalloc((void**)&dev_s0, block_N*info_tL*sizeof(float));
	hipMalloc((void**)&dev_p0, block_N*info_tL*sizeof(float));
	hipMalloc((void**)&dev_s1, block_N*info_tL*sizeof(float));
	hipMalloc((void**)&dev_p1, block_N*info_tL*sizeof(float));

	/*copy s/p from host to device*/
	float *upper_s = new float[info_tL];
	float *upper_p = new float[info_tL];
	float *lower_s = new float[info_tL];
	float *lower_p = new float[info_tL];
	switch (CQI)
	{
	case 0:
	case 1:
	case 2:
	case 3:
	case 12:
	{
		depuncture1(upper_s, upper_p, lower_s, lower_p, input_c, info_len, turbo_code_L);
		break;
	}
	case 4:
	case 6:
	{
		depuncture2(upper_s, upper_p, lower_s, lower_p, input_c, info_len, turbo_code_L);
		break;
	}
	case 5:
	case 7:
	case 9:
	case 10:
	case 11:
	{
		depuncture3(upper_s, upper_p, lower_s, lower_p, input_c, info_len, turbo_code_L);
		break;
	}
	case 8:
	{
		depuncture4(upper_s, upper_p, lower_s, lower_p, input_c, info_len, turbo_code_L);
		break;
	}
	default:
	{
		printf("CQI must between 0 and 11\n");
	}
	}
	interleaveOut(lower_s, upper_s, interleaver_table, info_len);

	for (int i = 0; i < block_N; i++)
	{
		hipMemcpy(dev_s0 + i*info_tL, upper_s, info_tL*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_p0 + i*info_tL, upper_p, info_tL*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_s1 + i*info_tL, lower_s, info_tL*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_p1 + i*info_tL, lower_p, info_tL*sizeof(float), hipMemcpyHostToDevice);
	}
	delete[] upper_p;
	delete[] upper_s;
	delete[] lower_p;
	delete[] lower_s;

	/*use one smx to handle 2 blocks each has N threads*/

	hipEventRecord(start, 0);			 //event record

	gammaKernel << <block_N, info_thread_tN >> > (dev_gamma, dev_s0, dev_p0, NULL);
	for (int iter = 0; iter < 6; iter++)
	{

		abKernel << <block_N, ab_thread_N >> >(dev_llr, dev_gamma, dev_ab, last_alfa0, last_beta0, dev_para, iter);

		extKernel << <block_N, info_thread_N >> >(dev_llr, dev_a, dev_s0, dev_inter, 0);

		gammaKernel << <block_N, info_thread_tN >> >(dev_gamma, dev_s1, dev_p1, dev_a);


		abKernel << <block_N, ab_thread_N >> >(dev_llr, dev_gamma, dev_ab, last_alfa1, last_beta1, dev_para, iter);
		if (iter < 5)
		{
			extKernel << <block_N, info_thread_N >> >(dev_llr, dev_a, dev_s1, dev_inter, 1);
			
			gammaKernel << <block_N, info_thread_tN >> > (dev_gamma, dev_s0, dev_p0, dev_a);
		}
	}
	dataKernel << <block_N, info_thread_N >> >(dev_data, dev_llr, dev_inter);

	hipEventRecord(stop, 0);				 //event record
	hipEventSynchronize(stop);
	hipEventElapsedTime(&duration, start, stop);

	hipMemcpy(detected_data, dev_data, info_len*sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_s0);
	hipFree(dev_p0);
	hipFree(last_alfa0);
	hipFree(last_alfa1);
	hipFree(last_beta0);
	hipFree(last_beta1);
	hipFree(dev_s1);
	hipFree(dev_p1);
	hipFree(dev_data);
	hipFree(dev_ab);
	hipFree(dev_gamma);
	hipFree(dev_a);
	hipFree(dev_llr);
	//(hipUnbindTexture(tex_inter);
	//(hipUnbindTexture(tex_para);
	hipFree(dev_inter);
	hipFree(dev_para);

	return duration;
}
